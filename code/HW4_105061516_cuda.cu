
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int INF = 10000000;
const int V = 10010;
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B);
int ceil(int a, int b);
void callP1(int r, int B);
void callP2(int r, int B, 
int *block_start_x, int *block_start_y, 
int *block_height, int *block_width);
__global__ void cal_Pone(int* Dist_ij);
__global__ void cal_Ptwo(int* Dist_ij, int* Dist_ik, int* Dist_kj);


int n, m;	// Number of vertices, edges
static int Dist[V][V];
//int* Dist;

int main(int argc, char* argv[])
{
	hipSetDevice(0);
	input(argv[1]);
	int B = atoi(argv[3]);
	if(B < 1)
		B = 1;
	else if(B > 32)// B^2 < max thread == 1024 
		B = 32;
	/*
	cudaDeviceProp prop;
    if(cudaGetDeviceProperties(&prop, 0) == cudaSuccess) 
	{
    	printf("cuda version = %d \n" , prop.major ) ;
		printf("maxThreadsPerBlock = %d \n" , prop.maxThreadsPerBlock ) ;
		printf("totalGlobalMem = %d \n" , prop.totalGlobalMem ) ;
		printf(" maxThreadsDim[3] = %d, %d, %d\n" , prop.maxThreadsDim[1], prop.maxThreadsDim[2] , prop.maxThreadsDim[3] ) ;
		printf(" maxGridSize[3] = %d, %d, %d\n" , prop.maxGridSize[1] , prop.maxGridSize[2] , prop.maxGridSize[3] ) ;
    }
	//cuda version: 2
	//maxThreadsPerBlock: 1024
	//totalGlobalMem: 2066153472
	//maxThreadsDim: 1024, 64, 65535
	//maxGridSize: 65535, 65535, 1301000
	//*/
	block_FW(B);
	
	
	output(argv[2]);
	

	return 0;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);
	
	

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (i == j)	Dist[i][j] = 0;
			else		Dist[i][j] = INF;
		}
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		--a, --b;
		Dist[a][b] = v;
	}
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
			else					fprintf(outfile, "%d ", Dist[i][j]);
		}
		fprintf(outfile, "\n");
	}		
}

int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void block_FW(int B)
{
	int round = ceil(n, B);
	
	
	for (int r = 0; r < round; ++r) {
		/* Phase 1*/
		//printf("[%d]This is callP1\n", r);
		callP1(r, B);
		
		
		/* Phase 2*/
		//printf("This is Phase 2\n");
		int block_start_x[4] = {r,		r, 0,	r + 1};
		int block_start_y[4] = {0,	r + 1, r,		r};
		int block_height[4] = {1, 				1, r,	round - r - 1};
		int block_width[4] = {r,	round - r - 1, 1, 				1};
		callP2(r, B, block_start_x, block_start_y,
			block_height, block_width);
		/* hase 3*/
		//printf("This is Phase 3\n");
		int block_start_x2[4] = {0, 	0,	r + 1, r + 1};
		int block_start_y2[4] = {0,	r + 1,		0, r + 1};
		int block_height2[4] = {r, r,	round - r -1,	round - r -1};
		int block_width2[4] = {r,	round - r -1, r,	round - r -1};
		callP2(r, B, block_start_x2, block_start_y2,
			block_height2, block_width2);
		//printf("End one turn\n");
	}

}

void putDistInArray(int round, int B, int bias_x, int bias_y, 
	int block_height, int block_width,
	int* Dist_ij, int* Dist_ik, int* Dist_kj){
		
	int shm_size = sizeof(int) * B * B;
	hipError_t err = hipHostMalloc(&Dist_ij, shm_size * block_height * block_width, hipHostMallocDefault);
	if (err != 0)	printf("malloc Dist_ij error\n");
	err = hipHostMalloc(&Dist_ik, shm_size * block_height, hipHostMallocDefault);
	if (err != 0)	printf("malloc Dist_ik error\n");
	err = hipHostMalloc(&Dist_kj, shm_size * block_width, hipHostMallocDefault);
	if (err != 0)	printf("malloc Dist_kj error\n");
	int ktmp, itmp, jtmp, j, k;
	int kbias = round * B;
	int end_x = block_height * B;
	int end_y = block_width * B;
	bias_x *= B;
	bias_y *= B;
	//為何要重新歷過一遍?比原本的還慢!!
	//因要解決IO問題，不想直接IO全部
	//真的有比較差??不是被"資源利用率"騙到?(IO全部VS處理完再IO部分)
	for(int i = 0; i < end_x; ++i){
		itmp = bias_x + i;
		if(itmp >= n)
			itmp = n - 1;
		for(j = 0; j < end_y; ++j){
			jtmp = bias_y + j;
			if(jtmp >= n)
				jtmp = n - 1;
			Dist_ij[i * end_y + j] = Dist[itmp][jtmp];
		}
		for(k = 0; k < B; ++k){
			ktmp = k + kbias;
			if(ktmp >= n)
				ktmp = n - 1;
			Dist_ik[i * B + k] = Dist[itmp][ktmp];
		}
	}
	for(int k = 0; k < B; ++k){
		ktmp = k + kbias;
		if(ktmp >= n)
			ktmp = n - 1;
		for(int j = 0; j < end_y; ++j){
			jtmp = bias_y + j;
			if(jtmp >= n)
				jtmp = n - 1;
			Dist_kj[k * end_y + j] = Dist[ktmp][jtmp];
		}
	}
}
void putToDist(int round, int B, int bias_x, int bias_y, 
	int block_height, int block_width,
	int* Dist_ij){
	
	int itmp, jtmp;
	int end_x = block_height * B;
	int end_y = block_width * B;
	bias_x *= B;
	bias_y *= B;
	for(int i = 0; i < end_x; ++i){
		itmp = bias_x + i;
		if(itmp >= n)
			break;
		for(int j = 0; j < end_y; ++j){
			jtmp = bias_y + j;
			if(jtmp >= n)
				break;
			Dist[itmp][jtmp] = Dist_ij[i * end_y + j];
		}
	}
}
void callP1(int r, int B){
	
	int shm_size = sizeof(int) * B * B;
	dim3 blocksPerGrif1(1, 1);
	dim3 threadsPerBlock(B, B);
	
	int *Dist_ij;
	//putDistInArray(r, B, r, r, 1, 1, Dist_ij, Dist_ik, Dist_kj);
		
//	cudaError_t err = cudaMallocHost(&Dist_ij, shm_size * 1 * 1);
//	if (err != 0)	printf("malloc Dist_ij error\n");
//	err = cudaMallocHost(&Dist_ik, shm_size * 1);
//	if (err != 0)	printf("malloc Dist_ik error\n");
//	err = cudaMallocHost(&Dist_kj, shm_size * 1);
//	if (err != 0)	printf("malloc Dist_kj error\n");
//	int ktmp, itmp, jtmp, j, k;
//	int kbias = r * B;
//	//為何要重新歷過一遍?比原本的還慢!!
//	//因要解決IO問題，不想直接IO全部
//	//真的有比較差??不是被"資源利用率"騙到?(IO全部VS處理完再IO部分)
//	for(int i = 0; i < B; ++i){
//		itmp = kbias + i;
//		if(itmp >= n)
//			itmp = n - 1;
//		for(j = 0; j < B; ++j){
//			jtmp = kbias + j;
//			if(jtmp >= n)
//				jtmp = n - 1;
//			Dist_ij[i * B + j] = Dist[itmp][jtmp];
//		}
//		for(k = 0; k < B; ++k){
//			ktmp = k + kbias;
//			if(ktmp >= n)
//				ktmp = n - 1;
//			Dist_ik[i * B + k] = Dist[itmp][ktmp];
//		}
//	}
//	for(int k = 0; k < B; ++k){
//		ktmp = k + kbias;
//		if(ktmp >= n)
//			ktmp = n - 1;
//		for(int j = 0; j < B; ++j){
//			jtmp = kbias + j;
//			if(jtmp >= n)
//				jtmp = n - 1;
//			Dist_kj[k * B + j] = Dist[ktmp][jtmp];
//		}
//	}
	
	hipError_t err = hipHostMalloc(&Dist_ij, shm_size * 1 * 1, hipHostMallocDefault);
	if (err != 0)	printf("malloc Dist_ij error\n");
	int itmp, jtmp;
	int bias_y = r * B;
	for(int i = 0; i < B; ++i){
		itmp = bias_y + i;
		if(itmp >= n)
			itmp = n - 1;
		for(int j = 0; j < B; ++j){
			jtmp = bias_y + j;
			if(jtmp >= n)
				jtmp = n - 1;
			Dist_ij[i * B + j] = Dist[itmp][jtmp];
		}
	}
	
	
	int *Dist_ijg;
	
	//step 1: declare
	hipMalloc((void **)&Dist_ijg, shm_size);
	//step 2: copy
	hipMemcpy(Dist_ijg, Dist_ij, shm_size, hipMemcpyHostToDevice);
	
	cal_Pone<<< blocksPerGrif1 , threadsPerBlock , shm_size>>> 
		(Dist_ijg);
	//step 3: get return
	hipMemcpy(Dist_ij, Dist_ijg, shm_size, hipMemcpyDeviceToHost);
	//step 4: free gpu
	hipFree(Dist_ijg);
	putToDist(r, B, r, r, 1, 1, Dist_ij);
	
	hipHostFree(Dist_ij);
	
}
void callP2(int r, int B, 
int *block_start_x, int *block_start_y, 
int *block_height, int *block_width){
	
	int shm_size = sizeof(int) * B * B;
	dim3 threadsPerBlock(B, B);
	const int str_num = 4;
	hipStream_t stream[str_num];
	for(int i = 0; i < str_num; i++)
		hipStreamCreate(&stream[i]);
	int *Dist_all[str_num *6];// pointer array
	hipError_t err;
	int ktmp, itmp, jtmp, i2, j, k, bias_x, bias_y, end_x, end_y;
	int kbias = r * B;
	for(int i = 0; i < str_num; ++i){
		if( block_height[i] == 0 || block_width[i] == 0)
			continue;
		dim3 blocksPerGrif1( block_height[i], block_width[i]);
		int *Dist_ij2, *Dist_ik2, *Dist_kj2;
		//putDistInArray(r, B, block_start_x[i], block_start_y[i], 
		//	block_height[i], block_width[i], Dist_ij2, Dist_ik2, Dist_kj2);
		
		err = hipHostMalloc(&Dist_ij2, shm_size * block_height[i] * block_width[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_ij2 error\n");
		err = hipHostMalloc(&Dist_ik2, shm_size * block_height[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_ik2 error\n");
		err = hipHostMalloc(&Dist_kj2, shm_size * block_width[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_kj2 error\n");
		end_x = block_height[i] * B;
		end_y = block_width[i] * B;
		bias_x = block_start_x[i] *B;
		bias_y = block_start_y[i] *B;
		//為何要重新歷過一遍?比原本的還慢!!
		//因要解決IO問題，不想直接IO全部
		//真的有比較差??不是被"資源利用率"騙到?(IO全部VS處理完再IO部分)
		for(i2 = 0; i2 < end_x; ++i2){
			itmp = bias_x + i2;
			if(itmp >= n)
				itmp = n -1;
			for(j = 0; j < end_y; ++j){
				jtmp = bias_y + j;
				if(jtmp >= n)
					jtmp = n -1;
				Dist_ij2[i2 * end_y + j] = Dist[itmp][jtmp];
			}
			for(k = 0; k < B; ++k){
				ktmp = k + kbias;
				if(ktmp >= n)
					ktmp = n -1;
				Dist_ik2[i2 * B + k] = Dist[itmp][ktmp];
			}
		}
		for(int k = 0; k < B; ++k){
			ktmp = k + kbias;
			if(ktmp >= n)
				ktmp = n -1;
			for(int j = 0; j < end_y; ++j){
				jtmp = bias_y + j;
				if(jtmp >= n)
					jtmp = n -1;
				Dist_kj2[k * end_y + j] = Dist[ktmp][jtmp];
			}
		}
	
		Dist_all[i *6] = Dist_ij2;
		Dist_all[i *6 +1] = Dist_ik2;
		Dist_all[i *6 +2] = Dist_kj2;
		int *Dist_ijg2, *Dist_ikg2, *Dist_kjg2;
		//step 1: declare
		hipMalloc((void **)&Dist_ijg2, shm_size * block_height[i] * block_width[i]);
		hipMalloc((void **)&Dist_ikg2, shm_size * block_height[i]);
		hipMalloc((void **)&Dist_kjg2, shm_size * block_width[i]);
		Dist_all[i *6 +3] = Dist_ijg2;
		Dist_all[i *6 +4] = Dist_ikg2;
		Dist_all[i *6 +5] = Dist_kjg2;
		//step 2: copy
		hipMemcpyAsync(Dist_ijg2, Dist_ij2, shm_size * block_height[i] * block_width[i], 
			hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(Dist_ikg2, Dist_ik2, shm_size * block_height[i], 
			hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(Dist_kjg2, Dist_kj2, shm_size * block_width[i], 
			hipMemcpyHostToDevice, stream[i]);
	
		cal_Ptwo<<< blocksPerGrif1 , threadsPerBlock , shm_size, stream[i]>>> 
			(Dist_ijg2, Dist_ikg2, Dist_kjg2);
		//step 3: get return
		hipMemcpyAsync(Dist_ij2, Dist_ijg2, shm_size * block_height[i] * block_width[i], 
			hipMemcpyDeviceToHost, stream[i]);
	}
	//wait for stream
	for(int i = 0; i < str_num; i++){
		if( block_height[i] == 0 || block_width[i] == 0)
			continue;
		hipStreamSynchronize(stream[i]);
		putToDist(r, B, block_start_x[i], block_start_y[i],
			block_height[i], block_width[i], Dist_all[i *6]);
		//step 4: free gpu
		hipFree(Dist_all[i *6 +3]);
		hipFree(Dist_all[i *6 +4]);
		hipFree(Dist_all[i *6 +5]);
		hipHostFree(Dist_all[i *6]);
		hipHostFree(Dist_all[i *6 +1]);
		hipHostFree(Dist_all[i *6 +2]);
			
		hipStreamDestroy(stream[i]);
		
	}
}void callP2_v2(int r, int B, 
int *block_start_x, int *block_start_y, 
int *block_height, int *block_width){
	
	int shm_size = sizeof(int) * B * B;
	dim3 threadsPerBlock(B, B);
	const int str_num = 4;
	hipStream_t stream[str_num];
	for(int i = 0; i < str_num; i++)
		hipStreamCreate(&stream[i]);
	int *Dist_all[str_num *6];// pointer array
	hipError_t err;
	int ktmp, itmp, jtmp, i2, j, k, bias_x, bias_y, end_x, end_y;
	int kbias = r * B;
	for(int i = 0; i < str_num; ++i){
		dim3 blocksPerGrif1( block_height[i], block_width[i]);
		int *Dist_ij2, *Dist_ik2, *Dist_kj2;
		//putDistInArray(r, B, block_start_x[i], block_start_y[i], 
		//	block_height[i], block_width[i], Dist_ij2, Dist_ik2, Dist_kj2);
		
		err = hipHostMalloc(&Dist_ij2, shm_size * block_height[i] * block_width[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_ij2 error\n");
		err = hipHostMalloc(&Dist_ik2, shm_size * block_height[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_ik2 error\n");
		err = hipHostMalloc(&Dist_kj2, shm_size * block_width[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_kj2 error\n");
		end_x = block_height[i] * B;
		end_y = block_width[i] * B;
		bias_x = block_start_x[i] *B;
		bias_y = block_start_y[i] *B;
		//為何要重新歷過一遍?比原本的還慢!!
		//因要解決IO問題，不想直接IO全部
		//真的有比較差??不是被"資源利用率"騙到?(IO全部VS處理完再IO部分)
		for(i2 = 0; i2 < end_x; ++i2){
			itmp = bias_x + i2;
			if(itmp >= n)
				itmp = n -1;
			for(j = 0; j < end_y; ++j){
				jtmp = bias_y + j;
				if(jtmp >= n)
					jtmp = n -1;
				Dist_ij2[i2 * end_y + j] = Dist[itmp][jtmp];
			}
			for(k = 0; k < B; ++k){
				ktmp = k + kbias;
				if(ktmp >= n)
					ktmp = n -1;
				Dist_ik2[i2 * B + k] = Dist[itmp][ktmp];
			}
		}
		for(int k = 0; k < B; ++k){
			ktmp = k + kbias;
			if(ktmp >= n)
				ktmp = n -1;
			for(int j = 0; j < end_y; ++j){
				jtmp = bias_y + j;
				if(jtmp >= n)
					jtmp = n -1;
				Dist_kj2[k * end_y + j] = Dist[ktmp][jtmp];
			}
		}
	
		Dist_all[i *6] = Dist_ij2;
		Dist_all[i *6 +1] = Dist_ik2;
		Dist_all[i *6 +2] = Dist_kj2;
		int *Dist_ijg2, *Dist_ikg2, *Dist_kjg2;
		//step 1: declare
		hipMalloc((void **)&Dist_ijg2, shm_size * block_height[i] * block_width[i]);
		hipMalloc((void **)&Dist_ikg2, shm_size * block_height[i]);
		hipMalloc((void **)&Dist_kjg2, shm_size * block_width[i]);
		Dist_all[i *6 +3] = Dist_ijg2;
		Dist_all[i *6 +4] = Dist_ikg2;
		Dist_all[i *6 +5] = Dist_kjg2;
		//step 2: copy
		hipMemcpyAsync(Dist_ijg2, Dist_ij2, shm_size * block_height[i] * block_width[i], 
			hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(Dist_ikg2, Dist_ik2, shm_size * block_height[i], 
			hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(Dist_kjg2, Dist_kj2, shm_size * block_width[i], 
			hipMemcpyHostToDevice, stream[i]);
	
		cal_Ptwo<<< blocksPerGrif1 , threadsPerBlock , shm_size, stream[i]>>> 
			(Dist_ijg2, Dist_ikg2, Dist_kjg2);
		//step 3: get return
		hipMemcpyAsync(Dist_ij2, Dist_ijg2, shm_size * block_height[i] * block_width[i], 
			hipMemcpyDeviceToHost, stream[i]);
	}
	//wait for stream
	for(int i = 0; i < str_num; i++){
		hipStreamSynchronize(stream[i]);
		putToDist(r, B, block_start_x[i], block_start_y[i],
			block_height[i], block_width[i], Dist_all[i *6]);
		//step 4: free gpu
		hipFree(Dist_all[i *6 +3]);
		hipFree(Dist_all[i *6 +4]);
		hipFree(Dist_all[i *6 +5]);
		hipHostFree(Dist_all[i *6]);
		hipHostFree(Dist_all[i *6 +1]);
		hipHostFree(Dist_all[i *6 +2]);
			
		hipStreamDestroy(stream[i]);
		
	}
}

__global__ void cal_Pone(int* Dist_ij)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	extern __shared__ int DS[];
	int dsbias = threadIdx.x * blockDim.y + threadIdx.y;
	int offset_j = gridDim.y * blockDim.y;
	DS[dsbias] = Dist_ij[i * offset_j + j];//j range = blocksPerG.y
	__syncthreads();
	for (int k = 0; k < blockDim.x ; ++k) {//k range= B
		if (DS[i * blockDim.x + k] + DS[k * offset_j + j] < DS[dsbias])
			DS[dsbias] = DS[i * blockDim.x + k] + DS[k * offset_j + j];
		__syncthreads();
	}
	Dist_ij[i * offset_j + j] = DS[dsbias];// save value to shared memory
	__syncthreads();
	
}
__global__ void cal_Ptwo(int* Dist_ij, int* Dist_ik, int* Dist_kj)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	extern __shared__ int DS[];
	int dsbias = threadIdx.x * blockDim.y + threadIdx.y;
	int offset_j = gridDim.y * blockDim.y;
	DS[dsbias] = Dist_ij[i * offset_j + j];//j range = blocksPerG.y
	__syncthreads();
	for (int k = 0; k < blockDim.x ; ++k) {//k range= B
		if (Dist_ik[i * blockDim.x + k] + Dist_kj[k * offset_j + j] < DS[dsbias])
			DS[dsbias] = Dist_ik[i * blockDim.x + k] + Dist_kj[k * offset_j + j];
		Dist_ij[i * offset_j + j] = DS[dsbias];// save value to shared memory
		__syncthreads();
	}
}

/*
void cal(int B, int Round, 
	int block_start_x, int block_start_y, 
	int block_width, int block_height)
{
	int block_end_x = block_start_x + block_height;
	int block_end_y = block_start_y + block_width;

	for (int b_i =  block_start_x; b_i < block_end_x; ++b_i) {
		for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
			// To calculate B*B elements in the block (b_i, b_j)
			// For each block, it need to compute B times
			for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
				// To calculate original index of elements in the block (b_i, b_j)
				// For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
				int block_internal_start_x 	= b_i * B;
				int block_internal_end_x 	= (b_i +1) * B;
				int block_internal_start_y = b_j * B; 
				int block_internal_end_y 	= (b_j +1) * B;

				if (block_internal_end_x > n)	block_internal_end_x = n;
				if (block_internal_end_y > n)	block_internal_end_y = n;

				for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
					for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
						if (Dist[i * n + k] + Dist[k * n + j] < Dist[i * n + j])
							Dist[i * n + j] = Dist[i * n + k] + Dist[k * n + j];
					}
				}
			}
		}
	}
}
*/

