#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int INF = 10000000;
const int V = 10010;
void input(char *inFileName);
void output(char *outFileName);

void block_FW();
int ceil(int a, int b);
void callP1(int r);
void callP2(int r, 
int *block_start_x, int *block_start_y, 
int *block_height, int *block_width);
__global__ void cal_Pone(int* Dist_ij);
__global__ void cal_Ptwo(int* Dist_ij, int* Dist_ik, int* Dist_kj);


int n, m, B;	// Number of vertices, edges
static int Dist[V][V];
//int* Dist;

int main(int argc, char* argv[])
{
	hipSetDevice(0);
	input(argv[1]);
	B = atoi(argv[3]);
	if(B < 1)
		B = 1;
	else if(B > 32)
		B = 32;
	/*
	cudaDeviceProp prop;
    if(cudaGetDeviceProperties(&prop, 0) == cudaSuccess) 
	{
    	printf("cuda version = %d \n" , prop.major ) ;
		printf("maxThreadsPerBlock = %d \n" , prop.maxThreadsPerBlock ) ;
		printf("totalGlobalMem = %d \n" , prop.totalGlobalMem ) ;
		printf(" maxThreadsDim[3] = %d, %d, %d\n" , prop.maxThreadsDim[1], prop.maxThreadsDim[2] , prop.maxThreadsDim[3] ) ;
		printf(" maxGridSize[3] = %d, %d, %d\n" , prop.maxGridSize[1] , prop.maxGridSize[2] , prop.maxGridSize[3] ) ;
    }
	//cuda version: 2
	//maxThreadsPerBlock: 1024
	//totalGlobalMem: 2066153472
	//maxThreadsDim: 1024, 64, 65535
	//maxGridSize: 65535, 65535, 1301000
	//*/
	block_FW();
	
	
	output(argv[2]);
	

	return 0;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);
	
	

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (i == j)	Dist[i][j] = 0;
			else		Dist[i][j] = INF;
		}
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		--a, --b;
		Dist[a][b] = v;
	}
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
			else					fprintf(outfile, "%d ", Dist[i][j]);
		}
		fprintf(outfile, "\n");
	}		
}

int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void block_FW()
{
	int round = ceil(n, B);
	
	
	for (int r = 0; r < round; ++r) {
		/* Phase 1*/
		//printf("[%d]This is callP1\n", r);
		callP1(r);
		
		
		/* Phase 2*/
		//printf("This is Phase 2\n");
		int block_start_x[4] = {r,		r, 0,	r + 1};
		int block_start_y[4] = {0,	r + 1, r,		r};
		int block_height[4] = {1, 				1, r,	round - r - 1};
		int block_width[4] = {r,	round - r - 1, 1, 				1};
		callP2(r, block_start_x, block_start_y,
			block_height, block_width);
		/* hase 3*/
		//printf("This is Phase 3\n");
		int block_start_x2[4] = {0, 	0,	r + 1, r + 1};
		int block_start_y2[4] = {0,	r + 1,		0, r + 1};
		int block_height2[4] = {r, r,	round - r -1,	round - r -1};
		int block_width2[4] = {r,	round - r -1, r,	round - r -1};
		callP2(r, block_start_x2, block_start_y2,
			block_height2, block_width2);
		//printf("End one turn\n");
	}

}
void putInAij(int bias_y, int* Dist_ij){
		
	int i, j;
	bias_y *= B;
	//為何要重新歷過一遍?比原本的還慢!!
	//因要解決IO問題，不想直接IO全部
	//真的有比較差??不是被"資源利用率"騙到?(IO全部VS處理完再IO部分)

	int jlen = B;
	int itmp ;
	int jtmp ;
	// b+j > n
	if(bias_y + jlen > n)
		jlen = n - bias_y;
	// part 1: <, <
	for(i = 0; i < jlen; ++i){
		itmp = bias_y + i;
		for(j = 0; j < jlen; ++j){
			jtmp = bias_y + j;
			Dist_ij[i * B + j] = Dist[itmp][jtmp];
		}
	}
	itmp = (jlen -1) * B;
	// part 3: >, <=
	for(i = jlen; i < B; ++i)
		for(j = 0; j < jlen; ++j)
			Dist_ij[i * B + j] = Dist_ij[itmp + j];
	// part 2: >, >
	jtmp += jlen -1;//(jlen -1) *B + (jlen -1);
	for(i = jlen; i < B; ++i)
		for(j = jlen; j < B; ++j)
			Dist_ij[i * B + j] = Dist_ij[jtmp];
	// part 4: <=, >
	jtmp = (jlen -1);
	for(i = 0; i < jlen; ++i){
		itmp = i *B;
		for(j = jlen; j < B; ++j){
			Dist_ij[itmp + j] = Dist_ij[itmp + jtmp];
		}
	}
}
void putDistInArray_new(int round, int bias_x, int bias_y, 
	int block_height, int block_width,
	int* Dist_ij2, int* Dist_ik2, int* Dist_kj2){
		
	int itmp, jtmp, ktmp, i2, j, k;
	int kbias = round * B;
	int end_x = block_height * B;
	int end_y = block_width * B;
	bias_x *= B;
	bias_y *= B;
	//為何要重新歷過一遍?比原本的還慢!!
	//因要解決IO問題，不想直接IO全部
	//真的有比較差??不是被"資源利用率"騙到?(IO全部VS處理完再IO部分)
	int ilen = end_x;
	int jlen = end_y;
	int klen = B;
	// b+i2 > n
	if(bias_x + ilen > n)
		ilen = n - bias_x;
	// b+j > n
	if(bias_y + jlen > n)
		jlen = n - bias_y;
	// b+j > n
	if(kbias + klen > n)
		klen = n - kbias;
	
	// part 1: <, <, <
	for(i2 = 0; i2 < ilen; ++i2){
		itmp = bias_x + i2;
		for(j = 0; j < jlen; ++j){
			jtmp = bias_y + j;
			Dist_ij2[i2 * end_y + j] = Dist[itmp][jtmp];
		}
	}
	// part 5: <, <, <
	for(k = 0; k < klen; ++k){
		ktmp = k + kbias;
		for(j = 0; j < jlen; ++j){
			jtmp = bias_y + j;
			Dist_kj2[k * end_y + j] = Dist[ktmp][jtmp];
		}
		for(i2 = 0; i2 < ilen; ++i2){
			itmp = bias_x + i2;
			Dist_ik2[i2 * B + k] = Dist[itmp][ktmp];
		}
	}
	// part 5-2: <, X, > ||  X, <, >
	ktmp = klen - 1;
	jtmp = ktmp * end_y;
	for(k = klen; k < B; ++k){
		for(i2 = 0; i2 < ilen; ++i2){
			itmp = i2 * B;
			Dist_ik2[itmp+ k] = Dist_ik2[itmp + ktmp];
		}
		for(j = 0; j < jlen; ++j)
			Dist_kj2[k * end_y + j] = Dist_kj2[jtmp + j];
	}
	// part 3: >, <, X
	itmp = (ilen -1) *end_y;
	for(i2 = ilen; i2 < end_x; ++i2)
		for(j = 0; j < jlen; ++j)
			Dist_ij2[i2 * end_y + j] = Dist_ij2[itmp + j];
	// part 2: >, >, X
	itmp += jlen -1;//(ilen -1) * end_y + (jlen -1);
	for(i2 = ilen; i2 < end_x; ++i2)
		for(j = jlen; j < end_y; ++j)
			Dist_ij2[i2 * end_y + j] = Dist_ij2[itmp];
	
	// part 5-3: >, X, > || X, >, >
	itmp = (ilen -1) * B + (klen -1);
	jtmp += jlen -1;//(klen -1) * end_y + (jlen -1);
	for(k = klen; k < B; ++k){
		for(i2 = ilen; i2 < end_x; ++i2)
			Dist_ik2[i2 * B + k] = Dist_ik2[itmp];
		for(j = jlen; j < end_y; ++j)
			Dist_kj2[k * end_y + j] = Dist_kj2[jtmp];
	}
	// part 5-4: >, X, < || X, <, <
	itmp -= klen -1;//(ilen -1) * B ;
	jtmp = jlen -1;
	for(k = 0; k < klen; ++k){
		ktmp = k * end_y;
		for(i2 = ilen; i2 < end_x; ++i2)
			Dist_ik2[i2 * B + k] = Dist_ik2[itmp + k];
		for(j = jlen; j < end_y; ++j)
			Dist_kj2[ktmp + j] = Dist_kj2[ktmp + jtmp];
	}
	// part 4: <, >, X
	for(i2 = 0; i2 < ilen; ++i2){
		itmp = i2 * end_y;
		for(j = jlen; j < end_y; ++j)
			Dist_ij2[itmp + j] = Dist_ij2[itmp + jtmp];
	}
	
	
}

void putToDist(int round, int bias_x, int bias_y, 
	int block_height, int block_width,
	int* Dist_ij){
	
	int itmp, jtmp;
	int end_x = block_height * B;
	int end_y = block_width * B;
	int ilen = end_x;
	int jlen = end_y;
	bias_x *= B;
	bias_y *= B;
	if(end_x + bias_x > n)// if [ind_end+1] > n, change,if == ,no c
		ilen = n - bias_x;
	if(end_y + bias_y > n)
		jlen = n - bias_y;
	
	for(int i = 0; i < ilen; ++i){
		itmp = bias_x + i;
		for(int j = 0; j < jlen; ++j){
			jtmp = bias_y + j;
			Dist[itmp][jtmp] = Dist_ij[i * end_y + j];
		}
	}
}

void callP1(int r){
	
	int shm_size = sizeof(int) * B * B;
	dim3 blocksPerGrif1(1, 1);
	dim3 threadsPerBlock(B, B);
	
	int *Dist_ij;
	hipError_t err = hipHostMalloc(&Dist_ij, shm_size , hipHostMallocDefault);
	if (err != 0)	printf("malloc Dist_ij error\n");
	putInAij( r, Dist_ij);
	
	int *Dist_ijg;
	
	//step 1: declare
	hipMalloc((void **)&Dist_ijg, shm_size);
	//step 2: copy
	hipMemcpy(Dist_ijg, Dist_ij, shm_size, hipMemcpyHostToDevice);
	
	cal_Pone<<< blocksPerGrif1 , threadsPerBlock , shm_size>>> 
		(Dist_ijg);
	//step 3: get return
	hipMemcpy(Dist_ij, Dist_ijg, shm_size, hipMemcpyDeviceToHost);
	//step 4: free gpu
	hipFree(Dist_ijg);
	putToDist(r, r, r, 1, 1, Dist_ij);
	
	hipHostFree(Dist_ij);
	
}
void callP2(int r, 
int *block_start_x, int *block_start_y, 
int *block_height, int *block_width){
	
	int shm_size = sizeof(int) * B * B;
	dim3 threadsPerBlock(B, B);
	const int str_num = 4;
	hipStream_t stream[str_num];
	for(int i = 0; i < str_num; i++)
		hipStreamCreate(&stream[i]);
	int *Dist_all[str_num *6];// pointer array
	hipError_t err;
	for(int i = 0; i < str_num; ++i){
		if( block_height[i] == 0 || block_width[i] == 0)
			continue;
		dim3 blocksPerGrif1( block_height[i], block_width[i]);
		int *Dist_ij2, *Dist_ik2, *Dist_kj2;
		
		err = hipHostMalloc(&Dist_ij2, shm_size * block_height[i] * block_width[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_ij2 error\n");
		err = hipHostMalloc(&Dist_ik2, shm_size * block_height[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_ik2 error\n");
		err = hipHostMalloc(&Dist_kj2, shm_size * block_width[i], hipHostMallocDefault);
		if (err != 0)	printf("malloc Dist_kj2 error\n");
		putDistInArray_new(r, block_start_x[i], block_start_y[i], 
			block_height[i], block_width[i], Dist_ij2, Dist_ik2, Dist_kj2);
		
		Dist_all[i *6] = Dist_ij2;
		Dist_all[i *6 +1] = Dist_ik2;
		Dist_all[i *6 +2] = Dist_kj2;
		int *Dist_ijg2, *Dist_ikg2, *Dist_kjg2;
		//step 1: declare
		hipMalloc((void **)&Dist_ijg2, shm_size * block_height[i] * block_width[i]);
		hipMalloc((void **)&Dist_ikg2, shm_size * block_height[i]);
		hipMalloc((void **)&Dist_kjg2, shm_size * block_width[i]);
		Dist_all[i *6 +3] = Dist_ijg2;
		Dist_all[i *6 +4] = Dist_ikg2;
		Dist_all[i *6 +5] = Dist_kjg2;
		//step 2: copy
		hipMemcpyAsync(Dist_ijg2, Dist_ij2, shm_size * block_height[i] * block_width[i], 
			hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(Dist_ikg2, Dist_ik2, shm_size * block_height[i], 
			hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(Dist_kjg2, Dist_kj2, shm_size * block_width[i], 
			hipMemcpyHostToDevice, stream[i]);
	
		cal_Ptwo<<< blocksPerGrif1 , threadsPerBlock , shm_size, stream[i]>>> 
			(Dist_ijg2, Dist_ikg2, Dist_kjg2);
		//step 3: get return
		hipMemcpyAsync(Dist_ij2, Dist_ijg2, shm_size * block_height[i] * block_width[i], 
			hipMemcpyDeviceToHost, stream[i]);
	}
	//wait for stream
	for(int i = 0; i < str_num; i++){
		if( block_height[i] == 0 || block_width[i] == 0)
			continue;
		hipStreamSynchronize(stream[i]);
		putToDist(r, block_start_x[i], block_start_y[i],
			block_height[i], block_width[i], Dist_all[i *6]);
		//step 4: free gpu
		hipFree(Dist_all[i *6 +3]);
		hipFree(Dist_all[i *6 +4]);
		hipFree(Dist_all[i *6 +5]);
		hipHostFree(Dist_all[i *6]);
		hipHostFree(Dist_all[i *6 +1]);
		hipHostFree(Dist_all[i *6 +2]);
			
		hipStreamDestroy(stream[i]);
		
	}
}

__global__ void cal_Pone(int* Dist_ij)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	extern __shared__ int DS[];
	int dsbias = threadIdx.x * blockDim.y + threadIdx.y;
	int offset_j = gridDim.y * blockDim.y;
	DS[dsbias] = Dist_ij[i * offset_j + j];//j range = blocksPerG.y
	__syncthreads();
	for (int k = 0; k < blockDim.x ; ++k) {//k range= B
		if (DS[i * blockDim.x + k] + DS[k * offset_j + j] < DS[dsbias])
			DS[dsbias] = DS[i * blockDim.x + k] + DS[k * offset_j + j];
		__syncthreads();
	}
	Dist_ij[i * offset_j + j] = DS[dsbias];// save value to shared memory
	__syncthreads();
	
}
__global__ void cal_Ptwo(int* Dist_ij, int* Dist_ik, int* Dist_kj)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	extern __shared__ int DS[];
	int dsbias = threadIdx.x * blockDim.y + threadIdx.y;
	int offset_j = gridDim.y * blockDim.y;
	DS[dsbias] = Dist_ij[i * offset_j + j];//j range = blocksPerG.y
	__syncthreads();
	for (int k = 0; k < blockDim.x ; ++k) {//k range= B
		if (Dist_ik[i * blockDim.x + k] + Dist_kj[k * offset_j + j] < DS[dsbias])
			DS[dsbias] = Dist_ik[i * blockDim.x + k] + Dist_kj[k * offset_j + j];
	}
	Dist_ij[i * offset_j + j] = DS[dsbias];// save value to shared memory
	__syncthreads();
}

/*
void cal(int B, int Round, 
	int block_start_x, int block_start_y, 
	int block_width, int block_height)
{
	int block_end_x = block_start_x + block_height;
	int block_end_y = block_start_y + block_width;

	for (int b_i =  block_start_x; b_i < block_end_x; ++b_i) {
		for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
			// To calculate B*B elements in the block (b_i, b_j)
			// For each block, it need to compute B times
			for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
				// To calculate original index of elements in the block (b_i, b_j)
				// For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
				int block_internal_start_x 	= b_i * B;
				int block_internal_end_x 	= (b_i +1) * B;
				int block_internal_start_y = b_j * B; 
				int block_internal_end_y 	= (b_j +1) * B;

				if (block_internal_end_x > n)	block_internal_end_x = n;
				if (block_internal_end_y > n)	block_internal_end_y = n;

				for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
					for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
						if (Dist[i * n + k] + Dist[k * n + j] < Dist[i * n + j])
							Dist[i * n + j] = Dist[i * n + k] + Dist[k * n + j];
					}
				}
			}
		}
	}
}
*/

